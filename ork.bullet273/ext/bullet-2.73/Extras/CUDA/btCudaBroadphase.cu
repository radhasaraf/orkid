#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

#include <cstdlib>
#include <cstdio>
#include <string.h>

#include "cutil_math.h"
#include "hip/hip_math_constants.h"

#if defined(__APPLE__) || defined(MACOSX)
#include <GLUT/glut.h>
#else
#include <GL/glut.h>
#endif

#include <cuda_gl_interop.h>

#include "btCudaBroadphaseKernel.h"
//#include "radixsort.cu"


//----------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------
//               K E R N E L    F U N C T I O N S 
//----------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------

#ifdef __DEVICE_EMULATION__
	#define B_CUDA_USE_TEX 0
#else
	#define B_CUDA_USE_TEX 1
#endif

__device__ inline btCuda3F1U tex_fetch3F1U(float4 a) { return *((btCuda3F1U*)(&a)); }

#if B_CUDA_USE_TEX
	#define FETCH(t, i) tex_fetch3F1U(tex1Dfetch(t##Tex, i))
#else
	#define FETCH(t, i) t[i]
#endif

texture<uint2, 1, hipReadModeElementType> particleHashTex;
texture<uint, 1, hipReadModeElementType> cellStartTex;
texture<float4, 1, hipReadModeElementType> pAABBTex;

//----------------------------------------------------------------------------------------

__constant__ btCudaBroadphaseParams params;

//----------------------------------------------------------------------------------------

// calculate position in uniform grid
__device__ int3 btCuda_calcGridPos(float4 p)
{
    int3 gridPos;
    gridPos.x = floor((p.x - params.m_worldOriginX) / params.m_cellSizeX);
    gridPos.y = floor((p.y - params.m_worldOriginY) / params.m_cellSizeY);
    gridPos.z = floor((p.z - params.m_worldOriginZ) / params.m_cellSizeZ);
    return gridPos;
}

//----------------------------------------------------------------------------------------

// calculate address in grid from position (clamping to edges)
__device__ uint btCuda_calcGridHash(int3 gridPos)
{
    gridPos.x = max(0, min(gridPos.x, params.m_gridSizeX - 1));
    gridPos.y = max(0, min(gridPos.y, params.m_gridSizeY - 1));
    gridPos.z = max(0, min(gridPos.z, params.m_gridSizeZ - 1));
    return __mul24(__mul24(gridPos.z, params.m_gridSizeY), params.m_gridSizeX) + __mul24(gridPos.y, params.m_gridSizeX) + gridPos.x;
}

//----------------------------------------------------------------------------------------

// calculate grid hash value for each body using its AABB
__global__ void calcHashAABBD(btCuda3F1U* pAABB, uint2* pHash, uint numBodies)
{
    int index = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;
    if(index >= numBodies)
	{
		return;
	}
	btCuda3F1U bbMin = pAABB[index*2];
	btCuda3F1U bbMax = pAABB[index*2 + 1];
	float4 pos;
	pos.x = (bbMin.fx + bbMax.fx) * 0.5f;
	pos.y = (bbMin.fy + bbMax.fy) * 0.5f;
	pos.z = (bbMin.fz + bbMax.fz) * 0.5f;
    // get address in grid
    int3 gridPos = btCuda_calcGridPos(pos);
    uint gridHash = btCuda_calcGridHash(gridPos);
    // store grid hash and body index
    pHash[index] = make_uint2(gridHash, index);
}

//----------------------------------------------------------------------------------------

__global__ void findCellStartD(uint2* pHash, uint* cellStart, uint numBodies)
{
    int index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
    if(index >= numBodies)
	{
		return;
	}
    uint2 sortedData = pHash[index];
	// Load hash data into shared memory so that we can look 
	// at neighboring body's hash value without loading
	// two hash values per thread
	__shared__ uint sharedHash[257];
	sharedHash[threadIdx.x+1] = sortedData.x;
	if((index > 0) && (threadIdx.x == 0))
	{
		// first thread in block must load neighbor body hash
		volatile uint2 prevData = pHash[index-1];
		sharedHash[0] = prevData.x;
	}
	__syncthreads();
	if((index == 0) || (sortedData.x != sharedHash[threadIdx.x]))
	{
		cellStart[sortedData.x] = index;
	}
}

//----------------------------------------------------------------------------------------

__device__ uint cudaTestAABBOverlap(btCuda3F1U min0, btCuda3F1U max0, btCuda3F1U min1, btCuda3F1U max1)
{
	return	(min0.fx <= max1.fx)&& (min1.fx <= max0.fx) && 
			(min0.fy <= max1.fy)&& (min1.fy <= max0.fy) && 
			(min0.fz <= max1.fz)&& (min1.fz <= max0.fz); 
}

//----------------------------------------------------------------------------------------

__device__ void findPairsInCell(int3	gridPos,
								uint    index,
								uint2*  pHash,
								uint*   pCellStart,
								btCuda3F1U* pAABB, 
								uint*   pPairBuff,
								uint2*	pPairBuffStartCurr,
								uint	numBodies)
{
    if (	(gridPos.x < 0) || (gridPos.x > params.m_gridSizeX - 1)
		||	(gridPos.y < 0) || (gridPos.y > params.m_gridSizeY - 1)
		||  (gridPos.z < 0) || (gridPos.z > params.m_gridSizeZ - 1)) 
    {
		return;
	}
    uint gridHash = btCuda_calcGridHash(gridPos);
    // get start of bucket for this cell
    uint bucketStart = pCellStart[gridHash];
    if (bucketStart == 0xffffffff)
	{
        return;   // cell empty
	}
	// iterate over bodies in this cell
    uint2 sortedData = pHash[index];
	uint unsorted_indx = sortedData.y;
    btCuda3F1U min0 = FETCH(pAABB, unsorted_indx*2); 
	btCuda3F1U max0 = FETCH(pAABB, unsorted_indx*2 + 1);
	uint handleIndex =  min0.uw;
	uint2 start_curr = pPairBuffStartCurr[handleIndex];
	uint start = start_curr.x;
	uint curr = start_curr.y;
	uint2 start_curr_next = pPairBuffStartCurr[handleIndex+1];
	uint curr_max = start_curr_next.x - start - 1;
	uint bucketEnd = bucketStart + params.m_maxBodiesPerCell;
	bucketEnd = (bucketEnd > numBodies) ? numBodies : bucketEnd;
	for(uint index2 = bucketStart; index2 < bucketEnd; index2++) 
	{
        uint2 cellData = pHash[index2];
        if (cellData.x != gridHash)
        {
			break;   // no longer in same bucket
		}
		uint unsorted_indx2 = cellData.y;
        if (unsorted_indx2 < unsorted_indx) // check not colliding with self
        {   
			btCuda3F1U min1 = FETCH(pAABB, unsorted_indx2*2);
			btCuda3F1U max1 = FETCH(pAABB, unsorted_indx2*2 + 1);
			if(cudaTestAABBOverlap(min0, max0, min1, max1))
			{
				uint handleIndex2 = min1.uw;
				uint k;
				for(k = 0; k < curr; k++)
				{
					uint old_pair = pPairBuff[start+k] & (~BT_CUDA_PAIR_ANY_FLG);
					if(old_pair == handleIndex2)
					{
						pPairBuff[start+k] |= BT_CUDA_PAIR_FOUND_FLG;
						break;
					}
				}
				if(k == curr)
				{
					pPairBuff[start+curr] = handleIndex2 | BT_CUDA_PAIR_NEW_FLG;
					if(curr >= curr_max) 
					{ // not a good solution, but let's avoid crash
						break;
					}
					curr++;
				}
			}
		}
	}
	pPairBuffStartCurr[handleIndex] = make_uint2(start, curr);
    return;
}

//----------------------------------------------------------------------------------------

__global__ void
findOverlappingPairsD(	btCuda3F1U*	pAABB, uint2* pHash, uint* pCellStart, uint* pPairBuff, 
						uint2* pPairBuffStartCurr, uint numBodies)
{
    int index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
    if(index >= numBodies)
	{
		return;
	}
    uint2 sortedData = pHash[index];
	uint unsorted_indx = sortedData.y;
	btCuda3F1U bbMin = FETCH(pAABB, unsorted_indx*2);
	btCuda3F1U bbMax = FETCH(pAABB, unsorted_indx*2 + 1);
	float4 pos;
	pos.x = (bbMin.fx + bbMax.fx) * 0.5f;
	pos.y = (bbMin.fy + bbMax.fy) * 0.5f;
	pos.z = (bbMin.fz + bbMax.fz) * 0.5f;
    // get address in grid
    int3 gridPos = btCuda_calcGridPos(pos);
    // examine only neighbouring cells
    for(int z=-1; z<=1; z++) {
        for(int y=-1; y<=1; y++) {
            for(int x=-1; x<=1; x++) {
                findPairsInCell(gridPos + make_int3(x, y, z), index, pHash, pCellStart, pAABB, pPairBuff, pPairBuffStartCurr, numBodies);
            }
        }
    }
}

//----------------------------------------------------------------------------------------

__global__ void
findPairsLargeD(	btCuda3F1U* pAABB, uint2* pHash, uint* pCellStart, uint* pPairBuff, 
						uint2* pPairBuffStartCurr, uint numBodies, uint numLarge)
{
    int index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
    if(index >= numBodies)
	{
		return;
	}
    uint2 sortedData = pHash[index];
	uint unsorted_indx = sortedData.y;
	btCuda3F1U min0 = FETCH(pAABB, unsorted_indx*2);
	btCuda3F1U max0 = FETCH(pAABB, unsorted_indx*2 + 1);
	uint handleIndex =  min0.uw;
	uint2 start_curr = pPairBuffStartCurr[handleIndex];
	uint start = start_curr.x;
	uint curr = start_curr.y;
	uint2 start_curr_next = pPairBuffStartCurr[handleIndex+1];
	uint curr_max = start_curr_next.x - start - 1;
    for(uint i = 0; i < numLarge; i++)
    {
		uint indx2 = numBodies + i;
		btCuda3F1U min1 = FETCH(pAABB, indx2*2);
		btCuda3F1U max1 = FETCH(pAABB, indx2*2 + 1);
		if(cudaTestAABBOverlap(min0, max0, min1, max1))
		{
			uint k;
			uint handleIndex2 =  min1.uw;
			for(k = 0; k < curr; k++)
			{
				uint old_pair = pPairBuff[start+k] & (~BT_CUDA_PAIR_ANY_FLG);
				if(old_pair == handleIndex2)
				{
					pPairBuff[start+k] |= BT_CUDA_PAIR_FOUND_FLG;
					break;
				}
			}
			if(k == curr)
			{
				pPairBuff[start+curr] = handleIndex2 | BT_CUDA_PAIR_NEW_FLG;
				if(curr >= curr_max) 
				{ // not a good solution, but let's avoid crash
					break;
				}
				curr++;
			}
		}
    }
	pPairBuffStartCurr[handleIndex] = make_uint2(start, curr);
    return;
}

//----------------------------------------------------------------------------------------

__global__ void computePairCacheChangesD(uint* pPairBuff, uint2* pPairBuffStartCurr, uint* pPairScan, btCuda3F1U* pAABB, uint numBodies)
{
    int index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
    if(index >= numBodies)
	{
		return;
	}
	btCuda3F1U bbMin = pAABB[index * 2];
	uint handleIndex = bbMin.uw;
	uint2 start_curr = pPairBuffStartCurr[handleIndex];
	uint start = start_curr.x;
	uint curr = start_curr.y;
	uint *pInp = pPairBuff + start;
	uint num_changes = 0;
	for(uint k = 0; k < curr; k++, pInp++)
	{
		if(!((*pInp) & BT_CUDA_PAIR_FOUND_FLG))
		{
			num_changes++;
		}
	}
	pPairScan[index+1] = num_changes;
}

//----------------------------------------------------------------------------------------

__global__ void squeezeOverlappingPairBuffD(uint* pPairBuff, uint2* pPairBuffStartCurr, uint* pPairScan, uint* pPairOut, btCuda3F1U* pAABB, uint numBodies)
{
    int index = __mul24(blockIdx.x,blockDim.x) + threadIdx.x;
    if(index >= numBodies)
	{
		return;
	}
	btCuda3F1U bbMin = pAABB[index * 2];
	uint handleIndex = bbMin.uw;
	uint2 start_curr = pPairBuffStartCurr[handleIndex];
	uint start = start_curr.x;
	uint curr = start_curr.y;
	uint* pInp = pPairBuff + start;
	uint* pOut = pPairOut + pPairScan[index];
	uint* pOut2 = pInp;
	uint num = 0; 
	for(uint k = 0; k < curr; k++, pInp++)
	{
		if(!((*pInp) & BT_CUDA_PAIR_FOUND_FLG))
		{
			*pOut = *pInp;
			pOut++;
		}
		if((*pInp) & BT_CUDA_PAIR_ANY_FLG)
		{
			*pOut2 = (*pInp) & (~BT_CUDA_PAIR_ANY_FLG);
			pOut2++;
			num++;
		}
	}
	pPairBuffStartCurr[handleIndex] = make_uint2(start, num);
} // squeezeOverlappingPairBuffD()


//----------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------
//               E N D   O F    K E R N E L    F U N C T I O N S 
//----------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------
//----------------------------------------------------------------------------------------


//! Check for CUDA error
#  define CUT_CHECK_ERROR(errorMessage) do {                                 \
    hipError_t err = hipGetLastError();                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",    \
                errorMessage, __FILE__, __LINE__, hipGetErrorString( err) );\
        btCuda_exit(EXIT_FAILURE);                                           \
    }                                                                        \
    err = hipDeviceSynchronize();                                           \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error: %s in file '%s' in line %i : %s.\n",    \
                errorMessage, __FILE__, __LINE__, hipGetErrorString( err) );\
        btCuda_exit(EXIT_FAILURE);                                           \
    } } while (0)


#  define MY_CUDA_SAFE_CALL_NO_SYNC( call) do {                              \
    hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        btCuda_exit(EXIT_FAILURE);                                           \
    } } while (0)

#  define MY_CUDA_SAFE_CALL( call) do {                                      \
    MY_CUDA_SAFE_CALL_NO_SYNC(call);                                         \
    hipError_t err = hipDeviceSynchronize();                                 \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda errorSync in file '%s' in line %i : %s.\n",    \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        btCuda_exit(EXIT_FAILURE);                                           \
    } } while (0)


extern "C"
{

void btCuda_exit(int val)
{
	exit(val);
}

void btCuda_allocateArray(void** devPtr, unsigned int size)
{
    MY_CUDA_SAFE_CALL(hipMalloc(devPtr, size));
}

void btCuda_freeArray(void* devPtr)
{
    MY_CUDA_SAFE_CALL(hipFree(devPtr));
}

void btCuda_copyArrayFromDevice(void* host, const void* device, unsigned int size)
{   
    MY_CUDA_SAFE_CALL(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));
}

void btCuda_copyArrayToDevice(void* device, const void* host, unsigned int size)
{
    MY_CUDA_SAFE_CALL(hipMemcpy((char*)device, host, size, hipMemcpyHostToDevice));
}

void btCuda_setParameters(btCudaBroadphaseParams* hostParams)
{
    // copy parameters to constant memory
    MY_CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(btCudaBroadphaseParams)));
}

//Round a / b to nearest higher integer value
int btCuda_iDivUp(int a, int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

// compute grid and thread block size for a given number of elements
void btCuda_computeGridSize(int n, int blockSize, int &numBlocks, int &numThreads)
{
    numThreads = min(blockSize, n);
    numBlocks = btCuda_iDivUp(n, numThreads);
}

void btCuda_calcHashAABB(btCuda3F1U* pAABB, unsigned int* hash,	unsigned int numBodies)
{
    int numThreads, numBlocks;
    btCuda_computeGridSize(numBodies, 256, numBlocks, numThreads);
    // execute the kernel
    calcHashAABBD<<< numBlocks, numThreads >>>(pAABB, (uint2*)hash, numBodies);
    // check if kernel invocation generated an error
    CUT_CHECK_ERROR("calcHashAABBD kernel execution failed");
}

void btCuda_findCellStart(unsigned int* hash, unsigned int* cellStart, unsigned int numBodies, unsigned int numCells)
{
    int numThreads, numBlocks;
    btCuda_computeGridSize(numBodies, 256, numBlocks, numThreads);
	MY_CUDA_SAFE_CALL(hipMemset(cellStart, 0xffffffff, numCells*sizeof(uint)));
    findCellStartD<<< numBlocks, numThreads >>>((uint2*)hash, (uint*)cellStart, numBodies);
    CUT_CHECK_ERROR("Kernel execution failed: findCellStartD");
}

void btCuda_findOverlappingPairs(	btCuda3F1U*		pAABB, unsigned int* pHash,
									unsigned int*	pCellStart,
									unsigned int*	pPairBuff,
									unsigned int*	pPairBuffStartCurr,
									unsigned int	numBodies)
{
#if B_CUDA_USE_TEX
    MY_CUDA_SAFE_CALL(hipBindTexture(0, pAABBTex, pAABB, numBodies * 2 * sizeof(btCuda3F1U)));
#endif
    int numThreads, numBlocks;
    btCuda_computeGridSize(numBodies, 64, numBlocks, numThreads);
    findOverlappingPairsD<<< numBlocks, numThreads >>>(
		pAABB,
		(uint2*)pHash,
        (uint*)pCellStart,
		(uint*)pPairBuff,
		(uint2*)pPairBuffStartCurr,
		numBodies
	);
    CUT_CHECK_ERROR("Kernel execution failed: bt_CudaFindOverlappingPairsD");
#if B_CUDA_USE_TEX
    MY_CUDA_SAFE_CALL(hipUnbindTexture(pAABBTex));
#endif
 } // btCuda_findOverlappingPairs()



void btCuda_findPairsLarge(	btCuda3F1U*	pAABB, unsigned int* pHash,
							unsigned int*	pCellStart,
							unsigned int*	pPairBuff,
							unsigned int*	pPairBuffStartCurr,
							unsigned int	numBodies,
							unsigned int	numLarge)
{
#if B_CUDA_USE_TEX
    MY_CUDA_SAFE_CALL(hipBindTexture(0, pAABBTex, pAABB, (numBodies+numLarge) * 2 * sizeof(btCuda3F1U)));
#endif
    int numThreads, numBlocks;
    btCuda_computeGridSize(numBodies, 64, numBlocks, numThreads);
    findPairsLargeD<<< numBlocks, numThreads >>>(
		pAABB,
		(uint2*)pHash,
        (uint*)pCellStart,
		(uint*)pPairBuff,
		(uint2*)pPairBuffStartCurr,
		numBodies,
		numLarge
	);
    CUT_CHECK_ERROR("Kernel execution failed: btCuda_findPairsLargeD");
#if B_CUDA_USE_TEX
    MY_CUDA_SAFE_CALL(hipUnbindTexture(pAABBTex));
#endif
 } // btCuda_findPairsLarge()



void btCuda_computePairCacheChanges(unsigned int* pPairBuff, unsigned int* pPairBuffStartCurr, 
									unsigned int* pPairScan, btCuda3F1U* pAABB, unsigned int numBodies)
{
    int numThreads, numBlocks;
    btCuda_computeGridSize(numBodies, 256, numBlocks, numThreads);
    computePairCacheChangesD<<< numBlocks, numThreads >>>(
		(uint*)pPairBuff,
		(uint2*)pPairBuffStartCurr,
        (uint*)pPairScan,
        pAABB, 
        numBodies
	);
    CUT_CHECK_ERROR("Kernel execution failed: btCudaComputePairCacheChangesD");
 } // btCuda_computePairCacheChanges()


void btCuda_squeezeOverlappingPairBuff(	unsigned int* pPairBuff, unsigned int* pPairBuffStartCurr, unsigned int* pPairScan, 
										unsigned int* pPairOut, btCuda3F1U* pAABB, unsigned int numBodies)
{
    int numThreads, numBlocks;
    btCuda_computeGridSize(numBodies, 256, numBlocks, numThreads);
    squeezeOverlappingPairBuffD<<< numBlocks, numThreads >>>(
		(uint*)pPairBuff,
		(uint2*)pPairBuffStartCurr,
        (uint*)pPairScan,
        (uint*)pPairOut,
        pAABB, 
        numBodies
	);
    CUT_CHECK_ERROR("Kernel execution failed: btCudaSqueezeOverlappingPairBuffD");
} // btCuda_squeezeOverlappingPairBuff()


}   // extern "C"
